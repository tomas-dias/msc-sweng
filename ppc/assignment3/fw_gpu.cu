#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "workshop.h"

#define GRAPH_SIZE 2000

#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff

#define THREADS_PER_BLOCK_SIDE 16 //16 * 16* 1 = 256
#define BLOCKS_PER_GRAPH_SIDE ((GRAPH_SIZE+THREADS_PER_BLOCK_SIDE-1) / THREADS_PER_BLOCK_SIDE) // Baseado no utilizado para o problema dot-product

/*
void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        int r;
        r = rand() % 40;
        if (r > 20) {
          r = INF;
        }

        D(i, j) = r;
      }
    }
  }
}
*/

void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        D(i, j) = i * j;
        if (i % 2==0)
          D(i, j) += j * 4;
      }
    }
  }
}

__global__ void fw_gpu_kernel(int graph_size, int *output, int k) {

  int i = blockDim.y * blockIdx.y + threadIdx.y;
  int j = blockDim.x * blockIdx.x + threadIdx.x;

  if( ( (i < graph_size) && (j < graph_size) ) && ( D(i, k) + D(k, j) < D(i, j) ) )
    D(i, j) = D(i, k) + D(k, j);
}

void floyd_warshall_gpu(const int *graph, int graph_size, int *output) {

  int *dev_output;
  int size;

  size = sizeof(int) * graph_size * graph_size;
  HANDLE_ERROR(hipMalloc(&dev_output, size));
  hipMemcpy(dev_output, graph, size, hipMemcpyHostToDevice);

  dim3 dimGrid(BLOCKS_PER_GRAPH_SIDE, BLOCKS_PER_GRAPH_SIDE, 1);
  dim3 dimBlock(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE, 1);

  for(int k = 0; k < graph_size; k++)
    fw_gpu_kernel<<<dimGrid, dimBlock>>>(graph_size, dev_output, k);
  
  hipMemcpy(output, dev_output, size, hipMemcpyDeviceToHost);
  hipFree(dev_output);

}

void floyd_warshall_cpu(const int *graph, int graph_size, int *output) {
  int i, j, k;

  memcpy(output, graph, sizeof(int) * graph_size * graph_size);

  for (k = 0; k < graph_size; k++) {
    for (i = 0; i < graph_size; i++) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
        }
      }
    }
  }
}

int main(int argc, char **argv) {
#define TIMER_START() gettimeofday(&tv1, NULL)
#define TIMER_STOP()                                                           \
  gettimeofday(&tv2, NULL);                                                    \
  timersub(&tv2, &tv1, &tv);                                                   \
  time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

  struct timeval tv1, tv2, tv;
  float time_delta;

  int *graph, *output_cpu, *output_gpu;

  int size;
  size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

  graph = (int *)malloc(size);
  assert(graph);

  output_cpu = (int *)malloc(size);
  assert(output_cpu);
  memset(output_cpu, 0, size);

  output_gpu = (int *)malloc(size);
  assert(output_gpu);

  generate_random_graph(graph, GRAPH_SIZE);

  fprintf(stderr, "running on cpu...\n");
  TIMER_START();
  floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  fprintf(stderr, "running on gpu...\n");
  TIMER_START();
  floyd_warshall_gpu(graph, GRAPH_SIZE, output_gpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  if(memcmp(output_cpu, output_gpu, size) != 0)
    fprintf(stderr, "FAIL!\n");
  else
    fprintf(stderr, "SUCCESS!\n");
  
  free(graph);
  free(output_cpu);
  free(output_gpu);

  return 0;
}
